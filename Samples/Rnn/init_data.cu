#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "fp16_emu.h" 
#include "init_data.cuh"

template void initGPUData<half1>( half1* data, int numElements, half1 value );
template void initGPUData<float>( float* data, int numElements, float value );
template void initGPUData<double>( double* data, int numElements, double value );

// Kernel and launcher to initialize GPU data to some constant value

template <typename T_ELEM>
__global__
void initGPUData_ker(T_ELEM *data, int numElements, T_ELEM value) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numElements) {
        data[tid] = value;
    }
}

template <typename T_ELEM>
void initGPUData(T_ELEM *data, int numElements, T_ELEM value) {
    dim3 gridDim;
    dim3 blockDim;

    blockDim.x = 1024;
    gridDim.x  = (numElements + blockDim.x - 1) / blockDim.x;

    initGPUData_ker<<<gridDim, blockDim>>>(data, numElements, value);
}


