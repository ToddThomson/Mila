
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <stdexcept>
#include <memory>
#include <vector>

template <typename T>
struct CudaTraits;

template <>
struct CudaTraits<float> {
    static constexpr hipDataType Type = HIP_R_32F;
};

template <>
struct CudaTraits<__half> {
    static constexpr hipDataType Type = HIP_R_16F;
};

template <>
struct CudaTraits<__hip_bfloat16> {
    static constexpr hipDataType Type = HIP_R_16BF;
};

// Forward fused matmul + bias + GELU
template <typename T>
void launchFusedMatmulBiasGelu(
    const T* A,        // [M x K]
    const T* B,        // [K x N]
    const T* bias,     // [N]
    T* C,              // [M x N]
    size_t M, size_t K, size_t N,
    hipblasLtHandle_t ltHandle,
    hipStream_t stream )
{
    hipblasOperation_t opTranspose = HIPBLAS_OP_T;
    hipblasOperation_t opNonTranspose = HIPBLAS_OP_N;

    // Create matmul descriptor
    hipblasLtMatmulDesc_t matmulDesc = nullptr;
    hipblasLtMatrixLayout_t aDesc = nullptr, bDesc = nullptr, cDesc = nullptr;
    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_GELU_AUX_BIAS;

    hipblasLtMatmulDescCreate( &matmulDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F );
    hipblasLtMatmulDescSetAttribute( matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opNonTranspose, sizeof( opNonTranspose ) );
    hipblasLtMatmulDescSetAttribute( matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opNonTranspose, sizeof( opNonTranspose ) );
    hipblasLtMatmulDescSetAttribute( matmulDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof( epilogue ) );
    hipblasLtMatmulDescSetAttribute( matmulDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof( bias ) );

    // Matrix layout: row-major
    hipblasLtMatrixLayoutCreate( &aDesc, CudaTraits<T>::Type, M, K, K );
    hipblasLtMatrixLayoutCreate( &bDesc, CudaTraits<T>::Type, K, N, N );
    hipblasLtMatrixLayoutCreate( &cDesc, CudaTraits<T>::Type, M, N, N );

    float alpha = 1.0f;
    float beta = 0.0f;

    size_t workspaceSize = 1 << 22;  // 4MB
    void* workspace;
    hipMalloc( &workspace, workspaceSize );

    hipblasLtMatmulPreference_t preference;
    hipblasLtMatmulPreferenceCreate( &preference );
    hipblasLtMatmulPreferenceSetAttribute( preference,
        HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
        &workspaceSize,
        sizeof( workspaceSize ) );

    hipblasLtMatmulAlgo_t algo;
    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    hipblasLtMatmulAlgoGetHeuristic( ltHandle, matmulDesc,
        aDesc, bDesc, cDesc, cDesc,
        preference, 1, &heuristicResult, &returnedResults );

    if ( returnedResults == 0 ) {
        throw std::runtime_error( "No suitable cuBLASLt matmul algorithm found." );
    }

    algo = heuristicResult.algo;

    hipblasLtMatmul( ltHandle,
        matmulDesc,
        &alpha,
        A, aDesc,
        B, bDesc,
        &beta,
        C, cDesc,
        C, cDesc,
        &algo,
        workspace,
        workspaceSize,
        stream );

    hipFree( workspace );
    hipblasLtMatmulPreferenceDestroy( preference );
    hipblasLtMatrixLayoutDestroy( aDesc );
    hipblasLtMatrixLayoutDestroy( bDesc );
    hipblasLtMatrixLayoutDestroy( cDesc );
    hipblasLtMatmulDescDestroy( matmulDesc );
}
