#include "hip/hip_runtime.h"
/**
 * @file CudaMatMulFp32.cu
 * @brief CUDA implementation of optimized matrix multiplication operations.
 *
 * This file contains CUDA kernels and host functions for high-performance matrix
 * multiplication operations, designed specifically for deep learning workloads.
 * The implementation utilizes shared memory and float4 vector operations for
 * improved memory bandwidth and computational efficiency.
 */

#include <hip/hip_runtime.h>
#include ""
#include "../CudaUtils.h"

namespace Mila::Dnn::Compute
{
    /**
     * @brief Scalar CUDA kernel for matrix multiplication with optional bias addition.
     *
     * This kernel computes the matrix multiplication Y = X * weight + bias without using vector operations,
     * making it suitable for matrices with dimensions that aren't multiples of 4. Each thread
     * computes one element of the output matrix.
     *
     * While less efficient than the vectorized version, this kernel works with any input dimensions
     * and serves as a fallback when the optimized kernel cannot be used.
     *
     * @param[out] Y Pointer to the output matrix of shape [B*T, OC]
     * @param[in] X Pointer to the input matrix of shape [B*T, C]
     * @param[in] weight Pointer to the weight matrix of shape [C, OC]
     * @param[in] bias Pointer to the bias vector (can be NULL if no bias is required)
     * @param[in] C Input feature dimension
     * @param[in] OC Output feature dimension
     */
    __global__ void matmul_forward_fp32_scalar_kernel(
        float* Y, const float* X, const float* weight, const float* bias, int C, int OC ) {

        // Calculate indices
        int row = blockIdx.x * blockDim.x + threadIdx.x; // B*T dimension
        int col = blockIdx.y * blockDim.y + threadIdx.y; // OC dimension

        // Check if we're within bounds
        if ( row < gridDim.x * blockDim.x && col < OC ) {
            float sum = 0.0f;

            // Compute dot product of input row and weight column
            for ( int i = 0; i < C; ++i ) {
                sum += X[ row * C + i ] * weight[ col * C + i ];
                //sum += X[row * C + i] * weight[i * OC + col];
            }

            // Add bias if provided
            if ( bias != NULL ) {
                sum += bias[ col ];
            }

            // Write result
            Y[ row * OC + col ] = sum;
        }
    }

    /**
     * @brief Loads a float4 vector from memory.
     *
     * This device function loads 4 adjacent float values from memory as a single float4 vector,
     * which helps optimize memory bandwidth utilization.
     *
     * @param[in] address Pointer to the memory location to load from (must be aligned properly)
     * @return float4 The loaded vector containing 4 float values
     */
    __device__ float4 ld_vec( const float* address ) {
        return *reinterpret_cast<const float4*>(address);
    }

    /**
     * @brief Stores a float4 vector to memory.
     *
     * This device function stores a float4 vector (4 adjacent float values) to memory,
     * which helps optimize memory bandwidth utilization.
     *
     * @param[out] address Pointer to the memory location to store to (must be aligned properly)
     * @param[in] val The float4 vector to store
     */
    __device__ void st_vec( float* address, float4 val ) {
        *reinterpret_cast<float4*>(address) = val;
    }

    /**
     * @brief Optimized CUDA kernel for matrix multiplication with optional bias addition.
     *
     * This kernel computes the matrix multiplication Y = X * weight + bias, where:
     * - X is the input matrix of shape [128 * blockIdx.x, C]
     * - weight is the weight matrix of shape [C, 128 * blockIdx.y]
     * - Y is the output matrix of shape [128 * blockIdx.x, 128 * blockIdx.y]
     * - bias is an optional bias vector of length 128 * blockIdx.y
     *
     * The kernel utilizes shared memory to cache chunks of input and weight matrices,
     * and each thread computes an 8x8 block of output elements. The implementation uses
     * float4 vector operations to improve memory bandwidth utilization.
     *
     * @param[out] Y Pointer to the output matrix
     * @param[in] X Pointer to the input matrix
     * @param[in] weight Pointer to the weight matrix
     * @param[in] bias Pointer to the bias vector (can be NULL if no bias is required)
     * @param[in] C Input feature dimension
     * @param[in] OC Output feature dimension
     *
     * @note This kernel requires that C and OC are multiples of 4 for proper alignment.
     * @note The kernel is configured with __launch_bounds__(256) to optimize occupancy.
     */
    __global__ void __launch_bounds__( 16 * 16 ) matmul_forward_fp32_vectorized_kernel(
        float* Y, const float* X, const float* weight, const float* bias, int C, int OC ) {
        // Each thread handles 8x8 elements; each block 128 by 128 elements.
        int oc = 8 * (blockIdx.y * blockDim.y + threadIdx.y);

        // buffers to cache chunks of the input matrices
        __shared__ float lhs_s[ 128 ][ 32 ];
        __shared__ float rhs_s[ 128 ][ 32 ];

        // adjust our pointers for the current block
        X += 128 * blockIdx.x * C;
        weight += 128 * blockIdx.y * C;
        Y += 128 * blockIdx.x * OC + 128 * blockIdx.y;

        // Initialize output values array, preloading with bias if available
        float vals[ 8 ][ 8 ] = {};
        if ( bias != NULL ) {
            for ( int i = 0; i < 8; i++ ) {
                for ( int j = 0; j < 8; j += 4 ) {
                    float4 b = ld_vec( bias + oc + j );
                    vals[ i ][ j + 0 ] = b.x;
                    vals[ i ][ j + 1 ] = b.y;
                    vals[ i ][ j + 2 ] = b.z;
                    vals[ i ][ j + 3 ] = b.w;
                }
            }
        }

        // Determine start index for shared memory loading
        int si_start = 4 * (16 * threadIdx.y + threadIdx.x);

        // Process the matrices in chunks of 32 columns
        for ( int so = 0; so < C; so += 32 ) {
            __syncthreads();
            // Collaborative loading of input and weight tiles into shared memory
            int xmod8 = threadIdx.x % 8;
            int xby8 = threadIdx.x / 8;
            int xo = 4 * xmod8;
            for ( int y = 2 * threadIdx.y + xby8; y < 128; y += 32 ) {
                st_vec( &lhs_s[ y ][ xo ], ld_vec( X + y * C + so + xo ) );
                st_vec( &rhs_s[ y ][ xo ], ld_vec( weight + y * C + so + xo ) );
            }
            __syncthreads();

            // Compute the matrix multiplication for this chunk
            for ( int si = si_start; si < si_start + 32; si += 4 ) {
                float4 rhs[ 8 ];
                for ( int u = 0; u < 8; ++u ) {
                    rhs[ u ] = ld_vec( &rhs_s[ u + 8 * threadIdx.y ][ si % 32 ] );
                }

                for ( int ii = 0; ii < 8; ++ii ) {
                    float4 lhs = ld_vec( &lhs_s[ ii + 8 * threadIdx.x ][ si % 32 ] );
                    for ( int ji = 0; ji < 8; ++ji ) {
                        vals[ ii ][ ji ] += lhs.x * rhs[ ji ].x;
                        vals[ ii ][ ji ] += lhs.y * rhs[ ji ].y;
                        vals[ ii ][ ji ] += lhs.z * rhs[ ji ].z;
                        vals[ ii ][ ji ] += lhs.w * rhs[ ji ].w;
                    }
                }
            }
        }

        // Write results back to global memory
        for ( int i = 0; i < 8; ++i ) {
            for ( int j = 0; j < 8; j += 4 ) {
                float4 result;
                result.x = vals[ i ][ j + 0 ];
                result.y = vals[ i ][ j + 1 ];
                result.z = vals[ i ][ j + 2 ];
                result.w = vals[ i ][ j + 3 ];
                st_vec( Y + (8 * threadIdx.x + i) * OC + 8 * threadIdx.y + j, result );
            }
        }
    };

    /**
     * @brief Host function to perform matrix multiplication on the GPU with automatic kernel selection.
     *
     * This function chooses between the optimized vectorized kernel and the scalar fallback kernel
     * based on the dimensions of the matrices. The vectorized kernel is used when C and OC are both
     * multiples of 4; otherwise, the scalar kernel is used.
     *
     * @param[out] Y Pointer to the output matrix (B, T, OC)
     * @param[in] X Pointer to the input matrix (B, T, C)
     * @param[in] weight Pointer to the weight matrix (C, OC)
     * @param[in] bias Pointer to the bias vector (OC), can be NULL if no bias is applied
     * @param[in] B Batch size
     * @param[in] T Sequence length
     * @param[in] C Number of input channels
     * @param[in] OC Number of output channels
     * @param[in] stream CUDA stream to use for the kernel launch
     */
    void cuda_matmul_forward_fp32(
        float* Y, const float* X, const float* weight, const float* bias,
        int B, int T, int C, int OC,
        hipStream_t stream ) {

        int sqrt_block_size = 16;

        // Use scalar kernel if dimensions don't meet requirements for optimized kernel
        if ( C % 4 != 0 || OC % 4 != 0 ||
            B * T < 128 || OC < 128 ||  // Size requirements
            C < 32 ) {                  // Minimum size for chunking
            // Use a safe implementation for non-aligned dimensions
            dim3 gridDim( ceil_div( B * T, sqrt_block_size ), ceil_div( OC, sqrt_block_size ) );
            dim3 blockDim( sqrt_block_size, sqrt_block_size );

            // Use the scalar kernel that doesn't rely on float4
            matmul_forward_fp32_scalar_kernel <<<gridDim, blockDim, 0, stream >>> (Y, X, weight, bias, C, OC);
        }
        else {
            // Use optimized kernel for aligned dimensions
            dim3 gridDim( ceil_div( B * T, 8 * sqrt_block_size ), ceil_div( OC, 8 * sqrt_block_size ) );
            dim3 blockDim( sqrt_block_size, sqrt_block_size );

            matmul_forward_fp32_vectorized_kernel << <gridDim, blockDim, 0, stream >> > (Y, X, weight, bias, C, OC);
        }

        cudaCheck( hipGetLastError() );
    }
}