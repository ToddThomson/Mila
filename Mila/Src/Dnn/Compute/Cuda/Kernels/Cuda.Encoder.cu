#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <cassert>
#include "Cuda.Utils.h"

__device__ inline float4 add_float4( const float4& a, const float4& b ) {
    return make_float4( a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w );
}

// use of float4 leads to using 128-bit LDG / STG instructions in SASS,
// very helpful in memory-bound kernels like encoder_forward
__global__ void encoder_forward_kernel3( float4* out,
    const int* inp, const float4* wte, const float4* wpe,
    int B, int T, int C ) {
    int C4 = C / 4;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int N = B * T * C4;
    if ( idx < N ) {
        int bt = idx / C4;
        int b = bt / T;
        int t = bt % T;
        int c4 = idx % C4;
        int ix = inp[ b * T + t ];
        out[ b * T * C4 + t * C4 + c4 ] = add_float4( wte[ ix * C4 + c4 ], wpe[ t * C4 + c4 ] );
    }
}

void cuda_encoder_forward( 
    float* out,
    const int* inp,
    const float* wte, const float* wpe,
    int B, int T, int C ) {
    assert( C % 4 == 0 );
    const int block_size = 512;
    const int N = B * T * C;
    const int grid_size = ceil_div( N / 4, block_size );
    
    encoder_forward_kernel3 << <grid_size, block_size >> > ((float4*)out, inp, (float4*)wte, (float4*)wpe, B, T, C);
    
    cudaCheck( hipGetLastError() );
}
